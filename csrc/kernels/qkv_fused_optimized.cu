#include "hip/hip_runtime.h"
/******************************************************************************
 * QKV Fusion Kernel - Optimized Implementation (Phase 2)
 * Based on FasterTransformer's approach:
 * 1. Single GEMM with concatenated QKV weights
 * 2. Fused split + bias + transpose kernel
 ******************************************************************************/

 #include <cstdio>
 #include <hip/hip_runtime.h>
 #include <hip/hip_fp16.h>
 #include <hipblas.h>
 
 // CUTLASS includes for optimized GEMM
 // #include <cute/tensor.hpp>
 // #include <cutlass/cutlass.h>
 // #include <cutlass/numeric_types.h>
 // #include <cutlass/gemm/device/gemm.h>
 
 #include "../qkv_fused_params.h"
 
 namespace qkv_fusion {
 
 // using namespace cute;
 
 /******************************************************************************
  * Kernel 1: Fused QKV GEMM using CUTLASS
  * Input:  hidden_states [batch * seq_len, hidden_dim]
  * Weight: qkv_weight [hidden_dim, (num_q_heads + 2*num_kv_heads) * head_dim]
  * Output: qkv_buf [batch * seq_len, (num_q_heads + 2*num_kv_heads) * head_dim]
  * 
  * Matrix multiplication: qkv_buf = hidden_states @ qkv_weight
  * Dimensions: [M, K] @ [K, N] = [M, N]
  *   where M = batch_size * seq_len
  *         K = hidden_dim
  *         N = qkv_output_dim
  ******************************************************************************/
 
 #include <hipblas.h>
 #include <hip/hip_fp16.h>
 
// Optimized GEMM using cuBLAS with proper row-major handling
void launch_fused_qkv_gemm_cutlass(
    const half* hidden_states,      // [M, K] row-major
    const half* qkv_weight,         // [K, N] row-major
    half* qkv_buf,                  // [M, N] row-major
    int M,                          // batch_size * seq_len
    int N,                          // qkv_output_dim
    int K,                          // hidden_dim
    hipblasHandle_t cublas_handle,
    hipStream_t stream
) {
    const half alpha = __float2half(1.0f);
    const half beta = __float2half(0.0f);
    
    hipblasSetStream(cublas_handle, stream);
    
    // Row-major C = A @ B becomes column-major C^T = B^T @ A^T
    // We want: qkv_buf[M, N] = hidden_states[M, K] @ qkv_weight[K, N]
    // In column-major view: qkv_buf^T[N, M] = qkv_weight^T[N, K] @ hidden_states^T[K, M]
    //
    // cuBLAS call: C = alpha * op(A) * op(B) + beta * C
    // where A = qkv_weight^T, B = hidden_states^T, C = qkv_buf^T
    //
    // Since our data is row-major, we interpret it as transposed column-major:
    // - qkv_weight[K, N] row-major = qkv_weight^T[N, K] column-major
    // - hidden_states[M, K] row-major = hidden_states^T[K, M] column-major
    
    hipblasStatus_t status = hipblasGemmEx(
        cublas_handle,
        HIPBLAS_OP_N,        // Don't transpose qkv_weight^T (already transposed by row-major)
        HIPBLAS_OP_N,        // Don't transpose hidden_states^T (already transposed by row-major)
        N,                  // Rows of qkv_weight^T[N, K]
        M,                  // Columns of hidden_states^T[K, M]
        K,                  // Inner dimension
        &alpha,
        qkv_weight,         // First matrix: qkv_weight^T[N, K] in column-major view
        HIP_R_16F,
        N,                  // Leading dimension (rows in column-major = N)
        // K,
        hidden_states,      // Second matrix: hidden_states^T[K, M] in column-major view
        HIP_R_16F,
        K,                  // Leading dimension (rows in column-major = K)
        &beta,
        qkv_buf,            // Output: qkv_buf^T[N, M] in column-major view
        HIP_R_16F,
        N,                  // Leading dimension (rows in column-major = N)
        HIPBLAS_COMPUTE_16F,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP
    );
    
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("cuBLAS GEMM failed with status: %d\n", status);
    }
}
 
/******************************************************************************
 * Kernel 2: Split QKV + Add Bias + Transpose (Optimized)
 * 
 * Optimization: Process in chunks to improve memory coalescing and reduce branching
 * 
 * Input:  qkv_buf [batch * seq_len, (num_q_heads + 2*num_kv_heads) * head_dim]
 * Output: q_out [batch, num_q_heads, seq_len, head_dim]
 *         k_out [batch, num_kv_heads, seq_len, head_dim]
 *         v_out [batch, num_kv_heads, seq_len, head_dim]
 ******************************************************************************/

template<typename T>
__global__ void split_qkv_bias_transpose_kernel_optimized(
    T* __restrict__ q_out,
    T* __restrict__ k_out,
    T* __restrict__ v_out,
    const T* __restrict__ qkv_buf,
    const T* __restrict__ qkv_bias,
    const int batch_size,
    const int seq_len,
    const int num_q_heads,
    const int num_kv_heads,
    const int head_dim
) {
    const int token_num = batch_size * seq_len;
    const int q_size = num_q_heads * head_dim;
    const int kv_size = num_kv_heads * head_dim;
    
    // Grid-stride loop over tokens
    for (int token_idx = blockIdx.x; token_idx < token_num; token_idx += gridDim.x) {
        const int batch_id = token_idx / seq_len;
        const int seq_id = token_idx % seq_len;
        
        const T* qkv_row = qkv_buf + token_idx * (q_size + 2 * kv_size);
        
        // Process Q (coalesced access within warp)
        for (int i = threadIdx.x; i < q_size; i += blockDim.x) {
            const int head_id = i / head_dim;
            const int dim_id = i % head_dim;
            
            T val = qkv_row[i];
            if (qkv_bias != nullptr) {
                val = __hadd(val, qkv_bias[i]);
            }
            
            const int out_idx = batch_id * num_q_heads * seq_len * head_dim +
                               head_id * seq_len * head_dim +
                               seq_id * head_dim +
                               dim_id;
            q_out[out_idx] = val;
        }
        
        // Process K
        for (int i = threadIdx.x; i < kv_size; i += blockDim.x) {
            const int head_id = i / head_dim;
            const int dim_id = i % head_dim;
            
            T val = qkv_row[q_size + i];
            if (qkv_bias != nullptr) {
                val = __hadd(val, qkv_bias[q_size + i]);
            }
            
            const int out_idx = batch_id * num_kv_heads * seq_len * head_dim +
                               head_id * seq_len * head_dim +
                               seq_id * head_dim +
                               dim_id;
            k_out[out_idx] = val;
        }
        
        // Process V
        for (int i = threadIdx.x; i < kv_size; i += blockDim.x) {
            const int head_id = i / head_dim;
            const int dim_id = i % head_dim;
            
            T val = qkv_row[q_size + kv_size + i];
            if (qkv_bias != nullptr) {
                val = __hadd(val, qkv_bias[q_size + kv_size + i]);
            }
            
            const int out_idx = batch_id * num_kv_heads * seq_len * head_dim +
                               head_id * seq_len * head_dim +
                               seq_id * head_dim +
                               dim_id;
            v_out[out_idx] = val;
        }
    }
}
 
 /******************************************************************************
  * Host function: Orchestrates the two-kernel approach
  ******************************************************************************/
 
 void run_qkv_fusion_optimized(
     QKVFusedParams &params,
     hipblasHandle_t cublas_handle,
     hipStream_t stream
 ) {
     const int batch_size = params.batch_size;
     const int seq_len = params.seqlen;
     const int hidden_dim = params.hidden_dim;
     const int num_q_heads = params.num_q_heads;
     const int num_kv_heads = params.num_kv_heads;
     const int head_dim = params.head_dim;
     
     // Total output dimension: Q + K + V
     const int qkv_output_dim = (num_q_heads + 2 * num_kv_heads) * head_dim;
    const int token_num = batch_size * seq_len;
    const int M = token_num;
    const int N = qkv_output_dim;
    const int K = hidden_dim;
    
    // Use pre-allocated workspace (no hipMalloc overhead!)
    half* qkv_buf = reinterpret_cast<half*>(params.workspace_ptr);

    // Step 1: Single GEMM for all Q, K, V projections
     // hidden_states [M, K] @ qkv_weight [K, N] = qkv_buf [M, N]
     // where M = batch_size * seq_len
     //       K = hidden_dim
     //       N = (num_q_heads + 2*num_kv_heads) * head_dim
     //
     // For Qwen3: M = batch*seq, K = 3584, N = 5120
     
     launch_fused_qkv_gemm_cutlass(
         reinterpret_cast<const half*>(params.hidden_states_ptr),
         reinterpret_cast<const half*>(params.qkv_fused_weight_ptr),
         qkv_buf,
         M, N, K,
         cublas_handle,
         stream
     );
     
    // Step 2: Split QKV + Add Bias + Transpose (Optimized)
    // Use optimized kernel with better memory coalescing
    const int threads = 256;
    const int blocks = min(token_num, 512);  // Limit blocks for better occupancy
    
    split_qkv_bias_transpose_kernel_optimized<half><<<blocks, threads, 0, stream>>>(
        reinterpret_cast<half*>(params.q_out_ptr),
        reinterpret_cast<half*>(params.k_out_ptr),
        reinterpret_cast<half*>(params.v_out_ptr),
        qkv_buf,
        params.has_bias ? reinterpret_cast<const half*>(params.qkv_fused_bias_ptr) : nullptr,
        batch_size,
        seq_len,
        num_q_heads,
        num_kv_heads,
        head_dim
    );

    // No hipFree needed - workspace is managed by PyTorch!
    
    // Check for errors
     hipError_t err = hipGetLastError();
     if (err != hipSuccess) {
         printf("CUDA kernel error: %s\n", hipGetErrorString(err));
     }
 }
 
 /******************************************************************************
  * Alternative: Fully Fused Single-Kernel Approach (Advanced)
  * Combines GEMM + Split + Transpose in one kernel
  * Requires more complex implementation but eliminates intermediate buffer
  ******************************************************************************/
 
 template<int kBlockM, int kBlockN, int kHeadDim>
 __global__ void qkv_fusion_fully_fused_kernel(
     const half* __restrict__ hidden_states,
     const half* __restrict__ qkv_weight,
     const half* __restrict__ qkv_bias,
     half* __restrict__ q_out,
     half* __restrict__ k_out,
     half* __restrict__ v_out,
     const int batch_size,
     const int seq_len,
     const int hidden_dim,
     const int num_q_heads,
     const int num_kv_heads
 ) {
     // This is an advanced optimization that fuses everything into one kernel
     // Benefits:
     // - No intermediate buffer needed
     // - Better data locality
     // - Fewer kernel launches
     //
     // Implementation strategy:
     // 1. Each thread block loads a tile of hidden_states into shared memory
     // 2. Computes Q, K, V for that tile using shared memory
     // 3. Directly writes to final transposed output layout
     //
     // TODO: Implement in Phase 2.5 after basic optimization works
 }
 
 } // namespace qkv_fusion
 
 