#include "hip/hip_runtime.h"
/******************************************************************************
 * QKV Fusion Kernel - Optimized Implementation (Phase 2)
 * Based on FasterTransformer's approach:
 * 1. Single GEMM with concatenated QKV weights
 * 2. Fused split + bias + transpose kernel
 ******************************************************************************/

 #include <cstdio>
 #include <cstdint>
 #include <vector>
 #include <algorithm>
 #include <ATen/cuda/HIPContext.h>
 #include <hip/hip_runtime.h>
 #include <hip/hip_fp16.h>
 #include <hipblas.h>
 #include <hipblaslt.h>
  
  // CUTLASS includes for optimized GEMM
  // #include <cute/tensor.hpp>
  // #include <cutlass/cutlass.h>
  // #include <cutlass/numeric_types.h>
  // #include <cutlass/gemm/device/gemm.h>
  
  #include "../qkv_fused_params.h"
  
  namespace qkv_fusion {
  
  // using namespace cute;
  
 /******************************************************************************
  * Debug helpers for cuBLASLt setup
  ******************************************************************************/
 
 #ifndef LT_CHECK
 #define LT_CHECK(stmt)                                                                          \
     do {                                                                                        \
         hipblasStatus_t _lt_status = (stmt);                                                     \
         if (_lt_status != HIPBLAS_STATUS_SUCCESS) {                                              \
             printf("[LtErr] %s:%d status=%d\n", __FILE__, __LINE__, static_cast<int>(_lt_status));\
             return;                                                                             \
         }                                                                                       \
     } while (0)
 #endif
 
 #ifndef CUDA_CHECK
 #define CUDA_CHECK(stmt)                                                                         \
     do {                                                                                        \
         hipError_t _cuda_status = (stmt);                                                       \
         if (_cuda_status != hipSuccess) {                                                       \
             printf("[CudaErr] %s:%d status=%d (%s)\n", __FILE__, __LINE__,                      \
                    static_cast<int>(_cuda_status), hipGetErrorString(_cuda_status));           \
             return;                                                                             \
         }                                                                                       \
     } while (0)
 #endif
 
 static inline void dump_layout(const char* name, hipblasLtMatrixLayout_t desc) {
     uint32_t order = 0;
     int64_t rows = 0;
     int64_t cols = 0;
     int64_t ld = 0;
     size_t attr_size = 0;
     hipblasLtMatrixLayoutGetAttribute(desc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(order), &attr_size);
     hipblasLtMatrixLayoutGetAttribute(desc, HIPBLASLT_MATRIX_LAYOUT_ROWS, &rows, sizeof(rows), &attr_size);
     hipblasLtMatrixLayoutGetAttribute(desc, HIPBLASLT_MATRIX_LAYOUT_COLS, &cols, sizeof(cols), &attr_size);
     hipblasLtMatrixLayoutGetAttribute(desc, HIPBLASLT_MATRIX_LAYOUT_LD, &ld, sizeof(ld), &attr_size);
     printf("[LAYOUT] %s order=%u rows=%lld cols=%lld ld=%lld\n",
            name,
            static_cast<unsigned>(order),
            static_cast<long long>(rows),
            static_cast<long long>(cols),
            static_cast<long long>(ld));
 }
 
 static inline void memset_fp16(void* ptr, size_t elems, uint8_t pattern = 0x00) {
     CUDA_CHECK(hipMemset(ptr, pattern, elems * sizeof(uint16_t)));
 }
 
 __global__ void add_bias_rowmajor_kernel(half* __restrict__ data,
                                          const half* __restrict__ bias,
                                          int M,
                                          int N) {
     int idx = blockIdx.x * blockDim.x + threadIdx.x;
     const int total = M * N;
     for (int linear = idx; linear < total; linear += blockDim.x * gridDim.x) {
         const int col = linear % N;
         data[linear] = __hadd(data[linear], bias[col]);
     }
 }
 
 static bool run_cublas_gemm_ex_with_bias(
     const half* hidden_states,
     const half* qkv_weight,
     const half* qkv_bias,
     half* qkv_buf,
     int M,
     int N,
     int K,
     hipStream_t stream
 ) {
     hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
     hipblasStatus_t status = hipblasSetStream(handle, stream);
     if (status != HIPBLAS_STATUS_SUCCESS) {
         printf("[cuBLAS] hipblasSetStream failed: %d\n", static_cast<int>(status));
         return false;
     }
 
     const float alpha = 1.0f;
     const float beta = 0.0f;
 
     status = hipblasGemmEx(
         handle,
         HIPBLAS_OP_N,               // B^T (handled by layout trick)
         HIPBLAS_OP_N,               // A^T
         N,                         // m
         M,                         // n
         K,                         // k
         &alpha,
         qkv_weight,
         HIP_R_16F,
         N,                         // lda
         hidden_states,
         HIP_R_16F,
         K,                         // ldb
         &beta,
         qkv_buf,
         HIP_R_16F,
         N,                         // ldc
         HIPBLAS_COMPUTE_32F,
         CUBLAS_GEMM_DEFAULT_TENSOR_OP);
 
     if (status != HIPBLAS_STATUS_SUCCESS) {
         printf("[cuBLAS] hipblasGemmEx failed: %d\n", static_cast<int>(status));
         return false;
     }
 
     if (qkv_bias != nullptr) {
         const int threads = 256;
         const int blocks = std::min(
             static_cast<int>((static_cast<long long>(M) * N + threads - 1) / threads),
             65535);
         add_bias_rowmajor_kernel<<<blocks, threads, 0, stream>>>(
             qkv_buf, qkv_bias, M, N);
     }
 
     return true;
 }
  
  /******************************************************************************
   * Kernel 1: Fused QKV GEMM using CUTLASS
   * Input:  hidden_states [batch * seq_len, hidden_dim]
   * Weight: qkv_weight [hidden_dim, (num_q_heads + 2*num_kv_heads) * head_dim]
   * Output: qkv_buf [batch * seq_len, (num_q_heads + 2*num_kv_heads) * head_dim]
   * 
   * Matrix multiplication: qkv_buf = hidden_states @ qkv_weight
   * Dimensions: [M, K] @ [K, N] = [M, N]
   *   where M = batch_size * seq_len
   *         K = hidden_dim
   *         N = qkv_output_dim
  ******************************************************************************/
 
 // Optimized GEMM using cuBLASLt with bias epilogue fusion
 void launch_fused_qkv_gemm_cutlass(
     const half* hidden_states,      // [M, K] row-major
     const half* qkv_weight,         // [K, N] row-major
     const half* qkv_bias,           // [N] or nullptr
     half* qkv_buf,                  // [M, N] row-major
     int M,                          // batch_size * seq_len
     int N,                          // qkv_output_dim
     int K,                          // hidden_dim
     hipblasLtHandle_t cublaslt_handle,
     hipStream_t stream
 ) {
     const float alpha = 1.0f;
     const float beta = 0.0f;
 
     const size_t totalElements = static_cast<size_t>(M) * static_cast<size_t>(N);
     memset_fp16(qkv_buf, totalElements);
 
     hipblasLtMatrixLayout_t aLayout = nullptr;
     hipblasLtMatrixLayout_t bLayout = nullptr;
     hipblasLtMatrixLayout_t cLayout = nullptr;
     hipblasLtMatrixLayout_t dLayout = nullptr;
     hipblasLtOrder_t rowOrder = HIPBLASLT_ORDER_ROW;
 
     bool layouts_ok = (hipblasLtMatrixLayoutCreate(&aLayout, HIP_R_16F, M, K, K) ==
                        HIPBLAS_STATUS_SUCCESS) &&
                       (hipblasLtMatrixLayoutCreate(&bLayout, HIP_R_16F, K, N, N) ==
                        HIPBLAS_STATUS_SUCCESS) &&
                       (hipblasLtMatrixLayoutCreate(&cLayout, HIP_R_16F, M, N, N) ==
                        HIPBLAS_STATUS_SUCCESS) &&
                       (hipblasLtMatrixLayoutCreate(&dLayout, HIP_R_16F, M, N, N) ==
                        HIPBLAS_STATUS_SUCCESS);
 
     if (!layouts_ok) {
         printf("[cuBLASLt] Failed to create matrix layouts\n");
     } else {
         hipblasLtMatrixLayoutSetAttribute(
             aLayout, HIPBLASLT_MATRIX_LAYOUT_ORDER, &rowOrder, sizeof(rowOrder));
         hipblasLtMatrixLayoutSetAttribute(
             bLayout, HIPBLASLT_MATRIX_LAYOUT_ORDER, &rowOrder, sizeof(rowOrder));
         hipblasLtMatrixLayoutSetAttribute(
             cLayout, HIPBLASLT_MATRIX_LAYOUT_ORDER, &rowOrder, sizeof(rowOrder));
         hipblasLtMatrixLayoutSetAttribute(
             dLayout, HIPBLASLT_MATRIX_LAYOUT_ORDER, &rowOrder, sizeof(rowOrder));
 
         dump_layout("A", aLayout);
         dump_layout("B", bLayout);
         dump_layout("C", cLayout);
         dump_layout("D", dLayout);
     }
 
     auto try_matmul = [&](bool fuse_bias, bool& bias_fused) -> bool {
         bias_fused = false;
 
         hipblasLtMatmulDesc_t matmulDesc = nullptr;
         hipblasStatus_t status =
             hipblasLtMatmulDescCreate(&matmulDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
         if (status != HIPBLAS_STATUS_SUCCESS) {
             printf("[cuBLASLt] MatmulDescCreate failed: %d\n", static_cast<int>(status));
             return false;
         }
 
         hipblasOperation_t opN = HIPBLAS_OP_N;
         hipblasLtMatmulDescSetAttribute(
             matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &opN, sizeof(opN));
         hipblasLtMatmulDescSetAttribute(
             matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opN, sizeof(opN));
 
         if (fuse_bias && qkv_bias != nullptr) {
             hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_BIAS;
             status = hipblasLtMatmulDescSetAttribute(
                 matmulDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
             if (status != HIPBLAS_STATUS_SUCCESS) {
                 printf("[cuBLASLt] Failed to set epilogue attribute: %d\n",
                        static_cast<int>(status));
                 hipblasLtMatmulDescDestroy(matmulDesc);
                 return false;
             }
             const void* bias_ptr = qkv_bias;
             status = hipblasLtMatmulDescSetAttribute(
                 matmulDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias_ptr, sizeof(bias_ptr));
             if (status != HIPBLAS_STATUS_SUCCESS) {
                 printf("[cuBLASLt] Failed to set bias pointer attribute: %d\n",
                        static_cast<int>(status));
                 hipblasLtMatmulDescDestroy(matmulDesc);
                 return false;
             }
             bias_fused = true;
         }
 
         hipblasLtMatmulPreference_t preference = nullptr;
         status = hipblasLtMatmulPreferenceCreate(&preference);
         if (status != HIPBLAS_STATUS_SUCCESS) {
             printf("[cuBLASLt] PreferenceCreate failed: %d\n", static_cast<int>(status));
             hipblasLtMatmulDescDestroy(matmulDesc);
             return false;
         }
 
         const size_t workspaceCaps[] = {
             static_cast<size_t>(64) * 1024 * 1024,
             static_cast<size_t>(16) * 1024 * 1024,
             0
         };
 
         constexpr int maxAlgos = 32;
         std::vector<hipblasLtMatmulHeuristicResult_t> heuristicResults(maxAlgos);
 
         bool launchSuccess = false;
         hipblasStatus_t matmulStatus = HIPBLAS_STATUS_SUCCESS;
 
         for (size_t workspaceCap : workspaceCaps) {
             status = hipblasLtMatmulPreferenceSetAttribute(
                 preference,
                 HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
                 &workspaceCap,
                 sizeof(workspaceCap));
             if (status != HIPBLAS_STATUS_SUCCESS) {
                 printf("[cuBLASLt] PreferenceSetAttribute failed: %d (cap %zu)\n",
                        static_cast<int>(status), workspaceCap);
                 continue;
             }
 
             int returnedResults = 0;
             status = hipblasLtMatmulAlgoGetHeuristic(
                 cublaslt_handle,
                 matmulDesc,
                 aLayout,
                 bLayout,
                 cLayout,
                 dLayout,
                 preference,
                 maxAlgos,
                 heuristicResults.data(),
                 &returnedResults);
 
             if (status == HIPBLAS_STATUS_NOT_SUPPORTED || returnedResults == 0) {
                 printf("[cuBLASLt] No algorithms for workspace cap %zu bytes\n", workspaceCap);
                 continue;
             }
             if (status != HIPBLAS_STATUS_SUCCESS) {
                 printf("[cuBLASLt] Heuristic query failed: %d (cap %zu)\n",
                        static_cast<int>(status), workspaceCap);
                 continue;
             }
 
             for (int algoIdx = 0; algoIdx < returnedResults; ++algoIdx) {
                 size_t workspaceSize = heuristicResults[algoIdx].workspaceSize;
                 void* workspace = nullptr;
                 if (workspaceSize > 0) {
                     hipError_t allocStatus = hipMalloc(&workspace, workspaceSize);
                     if (allocStatus != hipSuccess) {
                         printf("[cuBLASLt] Workspace alloc failed (%zu bytes) status=%d\n",
                                workspaceSize, static_cast<int>(allocStatus));
                         workspace = nullptr;
                         workspaceSize = 0;
                     }
                 }
 
                 printf("[cuBLASLt] Trying algo %d (bias=%d) workspace=%zu bytes\n",
                        algoIdx, static_cast<int>(bias_fused), workspaceSize);
 
                 matmulStatus = hipblasLtMatmul(
                     cublaslt_handle,
                     matmulDesc,
                     &alpha,
                     hidden_states,
                     aLayout,
                     qkv_weight,
                     bLayout,
                     &beta,
                     qkv_buf,
                     cLayout,
                     qkv_buf,
                     dLayout,
                     &heuristicResults[algoIdx].algo,
                     workspace,
                     workspaceSize,
                     stream);
 
                 if (workspace != nullptr) {
                     hipFree(workspace);
                 }
 
                 if (matmulStatus == HIPBLAS_STATUS_SUCCESS) {
                     launchSuccess = true;
                     printf("[cuBLASLt] Selected algo %d (bias=%d)\n",
                            algoIdx, static_cast<int>(bias_fused));
                     break;
                 } else {
                     printf("[cuBLASLt] Algo %d failed with status %d (bias=%d)\n",
                            algoIdx, static_cast<int>(matmulStatus), static_cast<int>(bias_fused));
                 }
             }
 
             if (launchSuccess) {
                 break;
             }
         }
 
         hipblasLtMatmulPreferenceDestroy(preference);
         if (!launchSuccess) {
             bias_fused = false;
         }
 
         hipblasLtMatmulDescDestroy(matmulDesc);
         return launchSuccess;
     };
 
     bool bias_fused = false;
     bool success = false;
     if (layouts_ok) {
         success = try_matmul(qkv_bias != nullptr, bias_fused);
 
         if (!success && qkv_bias != nullptr) {
             printf("[cuBLASLt] Retrying matmul without bias epilogue\n");
             success = try_matmul(false, bias_fused);
         }
 
         if (success && qkv_bias != nullptr && !bias_fused) {
             const int threads = 256;
             const int blocks = std::min(
                 static_cast<int>((static_cast<long long>(M) * N + threads - 1) / threads),
                 65535);
             add_bias_rowmajor_kernel<<<blocks, threads, 0, stream>>>(
                 qkv_buf, qkv_bias, M, N);
         }
     }
 
     if (!success) {
         printf("[cuBLASLt] Falling back to hipblasGemmEx path\n");
         bool cublas_ok = run_cublas_gemm_ex_with_bias(
             hidden_states,
             qkv_weight,
             qkv_bias,
             qkv_buf,
             M,
             N,
             K,
             stream);
         if (!cublas_ok) {
             printf("cuBLAS fallback also failed; output remains uninitialized\n");
         }
     }
 
     if (aLayout) hipblasLtMatrixLayoutDestroy(aLayout);
     if (bLayout) hipblasLtMatrixLayoutDestroy(bLayout);
     if (cLayout) hipblasLtMatrixLayoutDestroy(cLayout);
     if (dLayout) hipblasLtMatrixLayoutDestroy(dLayout);
 }
  
 /******************************************************************************
  * Kernel 2: Split QKV + Add Bias + Transpose (Optimized)
  * 
  * Optimization: Process in chunks to improve memory coalescing and reduce branching
  * 
  * Input:  qkv_buf [batch * seq_len, (num_q_heads + 2*num_kv_heads) * head_dim]
  * Output: q_out [batch, num_q_heads, seq_len, head_dim]
  *         k_out [batch, num_kv_heads, seq_len, head_dim]
  *         v_out [batch, num_kv_heads, seq_len, head_dim]
  ******************************************************************************/
 
 template<typename T>
 __global__ void split_qkv_bias_transpose_kernel_optimized(
     T* __restrict__ q_out,
     T* __restrict__ k_out,
     T* __restrict__ v_out,
     const T* __restrict__ qkv_buf,
     const T* __restrict__ qkv_bias,
     const int batch_size,
     const int seq_len,
     const int num_q_heads,
     const int num_kv_heads,
     const int head_dim
 ) {
     const int token_num = batch_size * seq_len;
     const int q_size = num_q_heads * head_dim;
     const int kv_size = num_kv_heads * head_dim;
     
     // Grid-stride loop over tokens
     for (int token_idx = blockIdx.x; token_idx < token_num; token_idx += gridDim.x) {
         const int batch_id = token_idx / seq_len;
         const int seq_id = token_idx % seq_len;
         
         const T* qkv_row = qkv_buf + token_idx * (q_size + 2 * kv_size);
         
         // Process Q (coalesced access within warp)
         for (int i = threadIdx.x; i < q_size; i += blockDim.x) {
             const int head_id = i / head_dim;
             const int dim_id = i % head_dim;
             
             T val = qkv_row[i];
             if (qkv_bias != nullptr) {
                 val = __hadd(val, qkv_bias[i]);
             }
             
             const int out_idx = batch_id * num_q_heads * seq_len * head_dim +
                                head_id * seq_len * head_dim +
                                seq_id * head_dim +
                                dim_id;
             q_out[out_idx] = val;
         }
         
         // Process K
         for (int i = threadIdx.x; i < kv_size; i += blockDim.x) {
             const int head_id = i / head_dim;
             const int dim_id = i % head_dim;
             
             T val = qkv_row[q_size + i];
             if (qkv_bias != nullptr) {
                 val = __hadd(val, qkv_bias[q_size + i]);
             }
             
             const int out_idx = batch_id * num_kv_heads * seq_len * head_dim +
                                head_id * seq_len * head_dim +
                                seq_id * head_dim +
                                dim_id;
             k_out[out_idx] = val;
         }
         
         // Process V
         for (int i = threadIdx.x; i < kv_size; i += blockDim.x) {
             const int head_id = i / head_dim;
             const int dim_id = i % head_dim;
             
             T val = qkv_row[q_size + kv_size + i];
             if (qkv_bias != nullptr) {
                 val = __hadd(val, qkv_bias[q_size + kv_size + i]);
             }
             
             const int out_idx = batch_id * num_kv_heads * seq_len * head_dim +
                                head_id * seq_len * head_dim +
                                seq_id * head_dim +
                                dim_id;
             v_out[out_idx] = val;
         }
     }
 }
  
  /******************************************************************************
   * Host function: Orchestrates the two-kernel approach
   ******************************************************************************/
  
 void run_qkv_fusion_optimized(
     QKVFusedParams &params,
     hipblasLtHandle_t cublaslt_handle,
     hipStream_t stream
 ) {
     const int batch_size = params.batch_size;
     const int seq_len = params.seqlen;
     const int hidden_dim = params.hidden_dim;
     const int num_q_heads = params.num_q_heads;
     const int num_kv_heads = params.num_kv_heads;
     const int head_dim = params.head_dim;
     
     // Total output dimension: Q + K + V
     const int qkv_output_dim = (num_q_heads + 2 * num_kv_heads) * head_dim;
    const int token_num = batch_size * seq_len;
    const int M = token_num;
    const int N = qkv_output_dim;
    const int K = hidden_dim;
    
    // Use pre-allocated workspace (no hipMalloc overhead!)
    half* qkv_buf = reinterpret_cast<half*>(params.workspace_ptr);
 
    // Step 1: Single cuBLASLt GEMM with bias epilogue fusion
     // hidden_states [M, K] @ qkv_weight [K, N] = qkv_buf [M, N]
     // Bias is fused in the GEMM epilogue (no separate kernel needed!)
     //
     // For Qwen3: M = batch*seq, K = 2048, N = 5120
     
     launch_fused_qkv_gemm_cutlass(
         reinterpret_cast<const half*>(params.hidden_states_ptr),
         reinterpret_cast<const half*>(params.qkv_fused_weight_ptr),
         params.has_bias ? reinterpret_cast<const half*>(params.qkv_fused_bias_ptr) : nullptr,
         qkv_buf,
         M, N, K,
         cublaslt_handle,
         stream
     );
     
     // Step 2: NO SPLIT KERNEL NEEDED!
     // Bias is already added by cuBLASLt epilogue
     // Python will do the reshape/slice (zero-copy operations)
     //
     // Output qkv_buf is now [batch*seq, 5120] with bias already applied
     // Python will:
     //   1. view as [batch, seq, 5120]
     //   2. slice into Q[batch, seq, 4096], K[batch, seq, 512], V[batch, seq, 512]
     //   3. view as [batch, seq, heads, head_dim]
     //   4. transpose to [batch, heads, seq, head_dim]
     //
     // All of these are zero-copy view operations except the final transpose!
     
     /* SPLIT KERNEL REMOVED - Now handled in Python
     const int threads = 256;
     const int blocks = min(token_num, 512);
     
     split_qkv_bias_transpose_kernel_optimized<half><<<blocks, threads, 0, stream>>>(
         reinterpret_cast<half*>(params.q_out_ptr),
         reinterpret_cast<half*>(params.k_out_ptr),
         reinterpret_cast<half*>(params.v_out_ptr),
         qkv_buf,
         nullptr,  // Bias already added by cuBLASLt
         batch_size,
         seq_len,
         num_q_heads,
         num_kv_heads,
         head_dim
     );
     */
 
    // No hipFree needed - workspace is managed by PyTorch!
     
     // Check for errors
      hipError_t err = hipGetLastError();
      if (err != hipSuccess) {
          printf("CUDA kernel error: %s\n", hipGetErrorString(err));
      }
  }
  
  /******************************************************************************
   * Alternative: Fully Fused Single-Kernel Approach (Advanced)
   * Combines GEMM + Split + Transpose in one kernel
   * Requires more complex implementation but eliminates intermediate buffer
   ******************************************************************************/
  
  template<int kBlockM, int kBlockN, int kHeadDim>
  __global__ void qkv_fusion_fully_fused_kernel(
      const half* __restrict__ hidden_states,
      const half* __restrict__ qkv_weight,
      const half* __restrict__ qkv_bias,
      half* __restrict__ q_out,
      half* __restrict__ k_out,
      half* __restrict__ v_out,
      const int batch_size,
      const int seq_len,
      const int hidden_dim,
      const int num_q_heads,
      const int num_kv_heads
  ) {
      // This is an advanced optimization that fuses everything into one kernel
      // Benefits:
      // - No intermediate buffer needed
      // - Better data locality
      // - Fewer kernel launches
      //
      // Implementation strategy:
      // 1. Each thread block loads a tile of hidden_states into shared memory
      // 2. Computes Q, K, V for that tile using shared memory
      // 3. Directly writes to final transposed output layout
      //
      // TODO: Implement in Phase 2.5 after basic optimization works
  }
  
  } // namespace qkv_fusion
  
  
 